
#include <hip/hip_runtime.h>
/*
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void SuperCategoryFMLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
	caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), top[depth_-1]->mutable_gpu_data());
	for(int i = 0; i < depth_-1; ++i)
		caffe_gpu_set(top[i]->count(), (Dtype)0., top[i]->mutable_gpu_data());

	for(int m = 0; m < M_; ++m) {
		for( int i = depth_-2; i >= 0; --i ) {
			Blob<Dtype> * tops = top[i];
			Blob<Dtype> * bottoms = top[i+1];

			int base_idx = base_index_per_level_[i];
			for(int j = 0; j < node_num_per_level_[i]; ++j) {
				Tree * node = serialized_tree_[base_idx + j];
				const std::vector<shared_ptr<Tree> >* children = node->GetChildren();

				Dtype * top_data = &tops->mutable_gpu_data()[tops->offset(m,node->GetLabel())];

				for(std::vector<shared_ptr<Tree> >::const_iterator it = children->begin(); it != children->end(); ++it) {
					int offset = bottoms->offset(m,(*it)->GetLabel());
					const Dtype * bottom_data = &bottoms->gpu_data()[offset];
					caffe_gpu_axpy(H_*W_,(Dtype)(1.),bottom_data,top_data);
				}

				caffe_gpu_scal(H_*W_,(Dtype)(1./children->size()),top_data);
			}
		}
	}
}

template <typename Dtype>
void SuperCategoryFMLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
	if( propagate_down[0] == false )
		return;

	for(int m = 0; m < M_; ++m) {
		for( int i = 0; i < depth_-1; ++i ) {
			Blob<Dtype> * tops = top[i];
			Blob<Dtype> * bottoms = top[i+1];

			int base_idx = base_index_per_level_[i];
			for(int j = 0; j < node_num_per_level_[i]; ++j) {
				Tree * node = serialized_tree_[base_idx + j];
				const std::vector<shared_ptr<Tree> >* children = node->GetChildren();
				const Dtype * top_diff = &tops->gpu_diff()[tops->offset(m,node->GetLabel())];
				for(std::vector<shared_ptr<Tree> >::const_iterator it = children->begin(); it != children->end(); ++it) {
					int offset = bottoms->offset(m,(*it)->GetLabel());
					Dtype * bottom_diff = &bottoms->mutable_gpu_diff()[offset];

					caffe_gpu_axpy(H_*W_,(Dtype)(1./children->size()),top_diff,bottom_diff);	
				}

			}
		}
	}
	caffe_copy(bottom[0]->count(), top[depth_-1]->gpu_diff(), bottom[0]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(SuperCategoryFMLayer);

}  // namespace caffe
*/
